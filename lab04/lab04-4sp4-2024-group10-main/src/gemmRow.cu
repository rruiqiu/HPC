
#include <hip/hip_runtime.h>
#include <iostream>

//col order
__global__ void matmul_single_row(int m, int n, int k, const float *A, const float *B, float *C) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread processes only valid rows
    if (row < m) {
        // Traverse through shared dimension (k) and columns of B (n)
        for (int j = 0; j < n; ++j) { // Iterate over columns of B
            float sum = 0.0;
            for (int l = 0; l < k; ++l) {
                sum += A[row * k + l] * B[l * n + j];
            }
            C[row * n + j] = sum;
        }
    }
}
float matmul_single_row_Wrapper(float* h_A, float* h_B, float* h_C, int m, int n, int k) {

    // Calculate sizes for matrices
    size_t size_A = m * k * sizeof(float); // Size of matrix A
    size_t size_B = k * n * sizeof(float); // Size of matrix B
    size_t size_C = m * n * sizeof(float); // Size of matrix C

    // Allocate memory on the device (GPU)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice); // Initialize C to 0 on device

    // Launch the kernel with enough blocks and threads
    int threadsPerBlock = 256;  //define how many threads per block within the cuda, max cuda support is 1024
    int blocksPerGrid = (m + threadsPerBlock-1) / threadsPerBlock;

    // CUDA events to measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0);
    // Launch the kernel
    matmul_single_row<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, d_A, d_B, d_C);


    // Record the stop event
    hipEventRecord(stop, 0);

    // Record stop event and synchronize
    hipEventSynchronize(stop);
    // cudaEventCreate(&stop);

    // Calculate elapsed time
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    // Copy the result matrix from device to host
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Cleanup: Destroy events and free device memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return elapsed; // Return elapsed time in milliseconds
}

__global__ void matmul_mutiple_row(int m, int n, int k, const float *A, const float *B, float *C,int tiling_size) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  // Global thread index

    int start_row = thread_id * tiling_size;           // Starting row for this thread
    int end_row = min(start_row + tiling_size, m);     // Ending row (exclusive)

    for (int row = start_row; row < end_row; ++row) {      // Iterate through assigned rows
        for (int j = 0; j < n; ++j) {                     // Iterate through columns of B
            float sum = 0.0;
            for (int l = 0; l < k; ++l) {                 // Shared dimension
                sum += A[row * k + l] * B[l * n + j];
            }
            C[row * n + j] = sum;                         // Write result to C
        }
    }
}



float matmul_mutiple_rows_Wrapper(float* h_A, float* h_B, float* h_C, int m, int n, int k, int tiling_size) {

    // Calculate sizes for matrices
    size_t size_A = m * k * sizeof(float); // Size of matrix A
    size_t size_B = k * n * sizeof(float); // Size of matrix B
    size_t size_C = m * n * sizeof(float); // Size of matrix C

    // Allocate memory on the device (GPU)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice); // Initialize C to 0 on device

    // Launch the kernel with enough blocks and threads
    int threadsPerBlock = 256;  //define how many threads per block within the cuda, max cuda support is 1024
    int rows_per_thread = tiling_size; //define how many rows each thread process

    int total_rows = (m + rows_per_thread - 1) / rows_per_thread;
    int blocksPerGrid = (total_rows + threadsPerBlock-1) / threadsPerBlock;

    // CUDA events to measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start, 0);

    // Launch the kernel
    matmul_mutiple_row<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, d_A, d_B, d_C,rows_per_thread);

    // Record stop event and synchronize
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    // Copy the result matrix from device to host
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Cleanup: Destroy events and free device memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // std::cout << elapsed;
    return elapsed; // Return elapsed time in milliseconds
}



__global__ void matmul_multiple_rowA_colB(int m, int n, int k, const float *A, const float *B, float *C,int tiling_size_row_A,int tiling_size_col_B) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  // Global thread index
    
    int col_per_thread = n / tiling_size_col_B;

    int start_row = thread_id / tiling_size_col_B;           // Starting row for this thread
    int start_col = (thread_id % tiling_size_col_B) * col_per_thread;         //define the start row
    
    int end_row = min(start_row + tiling_size_row_A, m);     // Ending row (exclusive)
    
    // int end_col = min(start_col + col_per_thread, n);
    int end_col = thread_id % tiling_size_col_B == tiling_size_col_B-1 ? n : start_col + col_per_thread; //handle the last columns of B


    // printf("Thread ID: %d, Start Row: %d, Start Col: %d\n", thread_id, start_row, start_col); 
    for (int row = start_row; row < end_row; ++row) {      // Iterate through assigned rows
        for (int j = start_col; j < end_col; ++j) {                     // Iterate through columns of B
            float sum = 0.0;
            for (int l = 0; l < k; ++l) {                 // Shared dimension
                sum += A[row * k + l] * B[l * n + j];
            }
            C[row * n + j] = sum;                         // Write result to C
        }
    }
}

float matmul_mutiple_rowA_colB_Wrapper(float* h_A, float* h_B, float* h_C, int m, int n, int k, int tiling_size_row_A,int tiling_size_col_B) {

    // Calculate sizes for matrices
    size_t size_A = m * k * sizeof(float); // Size of matrix A
    size_t size_B = k * n * sizeof(float); // Size of matrix B
    size_t size_C = m * n * sizeof(float); // Size of matrix C

    // Allocate memory on the device (GPU)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    // Copy input matrices from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice); // Initialize C to 0 on device

    // Calculate the total number of tiles
    int num_tiles_row = (m + tiling_size_row_A - 1) / tiling_size_row_A; // Number of row tiles
    int num_tiles_col = (n + tiling_size_col_B - 1) / tiling_size_col_B; // Number of column tiles
    int total_tiles = num_tiles_row * num_tiles_col;                     // Total number of tiles

    // Configure threads and blocks
    int threadsPerBlock = 256;  // Number of threads per block
    int blocksPerGrid = (total_tiles + threadsPerBlock - 1) / threadsPerBlock;



    // CUDA events to measure execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start, 0);


    // Launch the kernel
    matmul_multiple_rowA_colB<<<blocksPerGrid, threadsPerBlock>>>(m, n, k, d_A, d_B, d_C, tiling_size_row_A, tiling_size_col_B);

    // Record stop event and synchronize
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    // Copy the result matrix from device to host
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Cleanup: Destroy events and free device memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return elapsed; // Return elapsed time in milliseconds
}